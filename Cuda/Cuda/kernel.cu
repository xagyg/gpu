#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <math.h>

__global__ void sieveKernel(bool* d_prime, int chunkStart, int chunkEnd, int sq) {
    int p = blockDim.x * blockIdx.x + threadIdx.x + chunkStart;

    if (p > chunkEnd || p < 2 || p > sq) return; // Ensure p is within valid range

    if (p <= sq && d_prime[p]) {
        for (int i = p * p; i <= chunkEnd; i += p) {
            d_prime[i] = false; // Mark all multiples of p as not prime
        }
    }
}

void sieveOfEratosthenes(int n) {
    bool* h_prime = (bool*)malloc((n + 1) * sizeof(bool)); // Host array
    bool* d_prime; // Device array

    // Initialize all entries as true (prime)
    for (int i = 0; i <= n; i++)
        h_prime[i] = true;

    h_prime[0] = h_prime[1] = false;

    // Allocate memory on the GPU
    hipError_t err = hipMalloc((void**)&d_prime, (n + 1) * sizeof(bool));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the initialized prime array from the host to the device
    err = hipMemcpy(d_prime, h_prime, (n + 1) * sizeof(bool), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy (HostToDevice) failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int chunkSize = 1000000; // Adjusted chunk size
    int blockSize = 128;      // Define the block size
    int numChunks = (n + chunkSize - 1) / chunkSize;
    int sq = sqrt(n);

    for (int chunk = 0; chunk < numChunks; ++chunk) {
        int chunkStart = chunk * chunkSize + 2;
        int chunkEnd = min((chunk + 1) * chunkSize + 1, n);

        int numBlocks = (chunkEnd - chunkStart + blockSize - 1) / blockSize;

        // blockSize is the number of threads

        sieveKernel << <numBlocks, blockSize >> > (d_prime, chunkStart, chunkEnd, sq);

        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // Copy the result back to the host
    err = hipMemcpy(h_prime, d_prime, (n + 1) * sizeof(bool), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy (DeviceToHost) failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Print some prime numbers
    //for (int p = 2; p < 1000; p++) {
    //    if (h_prime[p]) {
    //        printf("%d ", p);
    //    }
    //}
    //printf("\n");

    // Free memory
    free(h_prime);
    hipFree(d_prime);
}

int main() {
    int n = 1000000000; // Large number to test

    printf("Calculating the prime numbers smaller than or equal to %d:\n", n);

    // Start timing
    clock_t start = clock();

    sieveOfEratosthenes(n);

    // End timing
    clock_t end = clock();

    // Calculate the elapsed time in seconds
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("\nTime taken: %.2f seconds\n", time_spent);

    return 0;
}
